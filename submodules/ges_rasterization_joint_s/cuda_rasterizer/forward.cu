#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "forward.h"
#include "auxiliary.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;

// Forward method for converting the input spherical harmonics
// coefficients of each Gaussian to a simple RGB color.
__device__ glm::vec3 computeColorFromSH(int idx, int deg, int max_coeffs, const glm::vec3* means, glm::vec3 campos, const float* shs)
{
	// The implementation is loosely based on code for 
	// "Differentiable Point-Based Radiance Fields for 
	// Efficient View Synthesis" by Zhang et al. (2022)
	glm::vec3 pos = means[idx];
	glm::vec3 dir = pos - campos;
	dir = dir / glm::length(dir);

	glm::vec3* sh = ((glm::vec3*)shs) + idx * max_coeffs;
	glm::vec3 result = SH_C0 * sh[0];

	if (deg > 0)
	{
		float x = dir.x;
		float y = dir.y;
		float z = dir.z;
		result = result - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

		if (deg > 1)
		{
			float xx = x * x, yy = y * y, zz = z * z;
			float xy = x * y, yz = y * z, xz = x * z;
			result = result +
				SH_C2[0] * xy * sh[4] +
				SH_C2[1] * yz * sh[5] +
				SH_C2[2] * (2.0f * zz - xx - yy) * sh[6] +
				SH_C2[3] * xz * sh[7] +
				SH_C2[4] * (xx - yy) * sh[8];

			if (deg > 2)
			{
				result = result +
					SH_C3[0] * y * (3.0f * xx - yy) * sh[9] +
					SH_C3[1] * xy * z * sh[10] +
					SH_C3[2] * y * (4.0f * zz - xx - yy) * sh[11] +
					SH_C3[3] * z * (2.0f * zz - 3.0f * xx - 3.0f * yy) * sh[12] +
					SH_C3[4] * x * (4.0f * zz - xx - yy) * sh[13] +
					SH_C3[5] * z * (xx - yy) * sh[14] +
					SH_C3[6] * x * (xx - 3.0f * yy) * sh[15];
			}
		}
	}
	result += 0.5f;
	return result;
}

// Compute a 2D-to-2D mapping matrix from a tangent plane into a image plane
// given a 2D gaussian parameters.
__device__ void compute_transmat(
	const float3& p_orig,
	const glm::vec2 scale,
	float mod,
	const glm::vec4 rot,
	const float* projmatrix,
	const float* viewmatrix,
	const int W,
	const int H, 
	glm::mat3 &T,
	float3 &normal
) {

	glm::mat3 R = quat_to_rotmat(rot);
	glm::mat3 S = scale_to_mat(scale, mod);
	glm::mat3 L = R * S;

	// center of Gaussians in the camera coordinate
	glm::mat3x4 splat2world = glm::mat3x4(
		glm::vec4(L[0], 0.0),
		glm::vec4(L[1], 0.0),
		glm::vec4(p_orig.x, p_orig.y, p_orig.z, 1)
	);

	glm::mat4 world2ndc = glm::mat4(
		projmatrix[0], projmatrix[4], projmatrix[8], projmatrix[12],
		projmatrix[1], projmatrix[5], projmatrix[9], projmatrix[13],
		projmatrix[2], projmatrix[6], projmatrix[10], projmatrix[14],
		projmatrix[3], projmatrix[7], projmatrix[11], projmatrix[15]
	);

	glm::mat3x4 ndc2pix = glm::mat3x4(
		glm::vec4(float(W) / 2.0, 0.0, 0.0, float(W-1) / 2.0),
		glm::vec4(0.0, float(H) / 2.0, 0.0, float(H-1) / 2.0),
		glm::vec4(0.0, 0.0, 0.0, 1.0)
	);

	T = glm::transpose(splat2world) * world2ndc * ndc2pix;
	normal = transformVec4x3({L[2].x, L[2].y, L[2].z}, viewmatrix);

}

// Computing the bounding box of the 2D Gaussian and its center
// The center of the bounding box is used to create a low pass filter
__device__ bool compute_aabb(
	glm::mat3 T, 
	float cutoff,
	float2& point_image,
	float2& extent
) {
	glm::vec3 t = glm::vec3(cutoff * cutoff, cutoff * cutoff, -1.0f); //f 
	float d = glm::dot(t, T[2] * T[2]); //a
	if (d == 0.0) return false;
	glm::vec3 f = (1 / d) * t;

	glm::vec2 p = glm::vec2(
		glm::dot(f, T[0] * T[2]),
		glm::dot(f, T[1] * T[2])
	);

	glm::vec2 h0 = p * p - 
		glm::vec2(
			glm::dot(f, T[0] * T[0]),
			glm::dot(f, T[1] * T[1])
		);

	glm::vec2 h = sqrt(max(glm::vec2(1e-4, 1e-4), h0));
	point_image = {p.x, p.y};
	extent = {h.x, h.y};
	return true;
}

// Perform initial steps for each Gaussian prior to rasterization.
template<int C>
__global__ void preprocessCUDA(int P, int D, int M,
	const float* orig_points,
	const glm::vec2* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	const float* transMat_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, int H,
	const float tan_fovx, const float tan_fovy,
	const float focal_x, const float focal_y,
	int* radii,
	float2* points_xy_image,
	float* depths,
	float* transMats,
	float* rgb,
	float4* normal_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;

	// Initialize radius and touched tiles to 0. If this isn't changed,
	// this Gaussian will not be processed further.
	radii[idx] = 0;
	tiles_touched[idx] = 0;

	// Perform near culling, quit if outside.
	float3 p_view;
	if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		return;
	
	// Compute transformation matrix
	glm::mat3 T;
	float3 normal;
	if (transMat_precomp == nullptr)
	{
		compute_transmat(((float3*)orig_points)[idx], scales[idx], scale_modifier, rotations[idx], projmatrix, viewmatrix, W, H, T, normal);
		float3 *T_ptr = (float3*)transMats;
		T_ptr[idx * 3 + 0] = {T[0][0], T[0][1], T[0][2]};
		T_ptr[idx * 3 + 1] = {T[1][0], T[1][1], T[1][2]};
		T_ptr[idx * 3 + 2] = {T[2][0], T[2][1], T[2][2]};
	} else {
		glm::vec3 *T_ptr = (glm::vec3*)transMat_precomp;
		T = glm::mat3(
			T_ptr[idx * 3 + 0], 
			T_ptr[idx * 3 + 1],
			T_ptr[idx * 3 + 2]
		);
		normal = make_float3(0.0, 0.0, 1.0);
	}

#if DUAL_VISIABLE
	float cos = -sumf3(p_view * normal);
	if (cos == 0) return;
	float multiplier = cos > 0 ? 1: -1;
	normal = multiplier * normal;
#endif

	float cutoff = 3.4f;

	// Compute center and radius
	float2 point_image;
	float radius;
	{
		float2 extent;
		bool ok = compute_aabb(T, cutoff, point_image, extent);
		if (!ok) return;
		radius = ceil(max(extent.x, extent.y));
	}

	uint2 rect_min, rect_max;
	getRect(point_image, radius, rect_min, rect_max, grid);
	if ((rect_max.x - rect_min.x) * (rect_max.y - rect_min.y) == 0)
		return;

	// Compute colors 
	if (colors_precomp == nullptr) {
		glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs);
		rgb[idx * C + 0] = result.x;
		rgb[idx * C + 1] = result.y;
		rgb[idx * C + 2] = result.z;
	}

	depths[idx] = p_view.z;
	radii[idx] = (int)radius;
	points_xy_image[idx] = point_image;
	normal_opacity[idx] = {normal.x, normal.y, normal.z, opacities[idx]};
	tiles_touched[idx] = (rect_max.y - rect_min.y) * (rect_max.x - rect_min.x);
}

// Main rasterization method. Collaboratively works on one tile per
// block, each thread treats one pixel. Alternates between fetching 
// and rasterizing data.
template <uint32_t CHANNELS>
__global__ void __launch_bounds__(BLOCK_X * BLOCK_Y)
renderCUDA(
	const uint2* __restrict__ ranges,
	const uint32_t* __restrict__ point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float* __restrict__ features,
	const float* __restrict__ transMats,
	const float4* __restrict__ normal_opacity,
	int* __restrict__ n_contrib,
	const float* __restrict__ bg_color,
	const float* __restrict__ mod_depth, //
	float* __restrict__ out_color,
	float* __restrict__ out_others)
{
	// Identify current tile and associated min/max pixel range.
	auto block = cg::this_thread_block();
	uint32_t horizontal_blocks = (W + BLOCK_X - 1) / BLOCK_X;
	uint2 pix_min = { block.group_index().x * BLOCK_X, block.group_index().y * BLOCK_Y };
	uint2 pix_max = { min(pix_min.x + BLOCK_X, W), min(pix_min.y + BLOCK_Y , H) };
	uint2 pix = { pix_min.x + block.thread_index().x, pix_min.y + block.thread_index().y };
	uint32_t pix_id = W * pix.y + pix.x;
	float2 pixf = { (float)pix.x, (float)pix.y};

	// Check if this thread is associated with a valid pixel or outside.
	bool inside = pix.x < W&& pix.y < H;
	// Done threads can help with fetching, but don't rasterize
	bool done = !inside;

	/*
	if (inside){
		if (pix.x%2==0 && pix.y%2 == 0) {
			pixf.x += 0.75; pixf.y += 0.25;
		}
		else if (pix.x%2==0 && pix.y%2 == 1) {
			pixf.x += 0.25; pixf.y += 0.25;
		}
		else if (pix.x%2==1 && pix.y%2 == 0) {
			pixf.x += 0.75; pixf.y += 0.75;
		}
		else if (pix.x%2==1 && pix.y%2 == 1) {
			pixf.x += 0.25; pixf.y += 0.75;
		}
		pixf.x -= 0.5f;
		pixf.y -= 0.5f;
	}
	*/

	// Load start/end range of IDs to process in bit sorted list.
	uint2 range = ranges[block.group_index().y * horizontal_blocks + block.group_index().x];
	const int rounds = ((range.y - range.x + BLOCK_SIZE - 1) / BLOCK_SIZE);
	int toDo = range.y - range.x;

	// Allocate storage for batches of collectively fetched data.
	__shared__ int collected_id[BLOCK_SIZE];
	__shared__ float4 collected_normal_opacity[BLOCK_SIZE];
	__shared__ float3 collected_Tu[BLOCK_SIZE];
	__shared__ float3 collected_Tv[BLOCK_SIZE];
	__shared__ float3 collected_Tw[BLOCK_SIZE];

	// Initialize helper variables
	uint32_t contributor = 0;
	float C[CHANNELS] = { 0 };

	float minDepth = 99999.0f;
	int closeGeoGlobalIdx = -1;

	// First, find the min depth Geo (Note: We ignore tex Gaussians depth sort, it might have no imapct)
	for (int i = 0; i < rounds; i++, toDo -= BLOCK_SIZE)
	{
		// End if entire block votes that it is done rasterizing
		int num_done = __syncthreads_count(done);
		if (num_done == BLOCK_SIZE)
			break;

		// Collectively fetch per-Gaussian data from global to shared
		int progress = i * BLOCK_SIZE + block.thread_rank();
		if (range.x + progress < range.y)
		{
			int coll_id = point_list[range.x + progress];
			collected_id[block.thread_rank()] = coll_id;
			collected_normal_opacity[block.thread_rank()] = normal_opacity[coll_id];
			collected_Tu[block.thread_rank()] = {transMats[9 * coll_id+0], transMats[9 * coll_id+1], transMats[9 * coll_id+2]};
			collected_Tv[block.thread_rank()] = {transMats[9 * coll_id+3], transMats[9 * coll_id+4], transMats[9 * coll_id+5]};
			collected_Tw[block.thread_rank()] = {transMats[9 * coll_id+6], transMats[9 * coll_id+7], transMats[9 * coll_id+8]};
		}
		block.sync();

		// Iterate over current batch
		for (int j = 0; !done && j < min(BLOCK_SIZE, toDo); j++)
		{
			// Keep track of current position in range
			contributor++;

			// Fisrt compute two homogeneous planes, See Eq. (8)
			const float3 Tu = collected_Tu[j];
			const float3 Tv = collected_Tv[j];
			const float3 Tw = collected_Tw[j];
			float3 k = pixf.x * Tw - Tu;
			float3 l = pixf.y * Tw - Tv;
			float3 p = cross(k, l);
			if (p.z == 0.0) continue;
			float2 s = {p.x / p.z, p.y / p.z};
			float rho3d = (s.x * s.x + s.y * s.y); 
			float depth, rho;
			
			rho = rho3d;
			depth = (s.x * Tw.x + s.y * Tw.y) + Tw.z;
			
			if (depth < near_n) continue;

			float power = -0.5f * rho;
			if (power > 0.0f)
				continue;

			float G = __expf(power);
			// when opac>1, we truncate the gaussian where G < 1.0f / 255.0f
			if (G < 1.0f / 255.0f)
				continue;

			float4 nor_o = collected_normal_opacity[j];
			if (depth < minDepth) {
				minDepth = depth;
				closeGeoGlobalIdx = collected_id[j];
			}
			
		}
	}

	// All threads that treat valid pixel write out their final
	// rendering data to the frame and auxiliary buffers.
	if (inside)
	{	
		float modDepth = 0;
		//////////////
		if (closeGeoGlobalIdx >= 0) {
			modDepth = mod_depth[closeGeoGlobalIdx];

			// pixel not covered by geom gs, its out depth is 0
			out_others[pix_id] = minDepth; // linear, in view space
		}
		//////////////
		// // pixel not covered by geom gs, its mod_depth is 999999
		out_others[pix_id + H*W] = minDepth + modDepth;
		out_others[pix_id + 2*H*W] = (float)closeGeoGlobalIdx;

		n_contrib[pix_id] = closeGeoGlobalIdx;
		if (closeGeoGlobalIdx < 0) {// no geo, so we can see bg
			for (int ch = 0; ch < CHANNELS; ch++)
				out_color[ch * H * W + pix_id] = bg_color[ch];
		}
		else 
			for (int ch = 0; ch < CHANNELS; ch++)
				out_color[ch * H * W + pix_id] = features[closeGeoGlobalIdx * CHANNELS + ch];
	}

}

template<int C>
__global__ void computeSHCUDA(int P, int D, int M,
	const float* orig_points,
	const float* shs,
	const glm::vec3* cam_pos,
	float* rgb) {

	auto idx = cg::this_grid().thread_rank();
	if (idx >= P)
		return;
	glm::vec3 result = computeColorFromSH(idx, D, M, (glm::vec3*)orig_points, *cam_pos, shs);
	rgb[idx * C + 0] = result.x;
	rgb[idx * C + 1] = result.y;
	rgb[idx * C + 2] = result.z;
}

void FORWARD::render(
	const dim3 grid, dim3 block,
	const uint2* ranges,
	const uint32_t* point_list,
	int W, int H,
	float focal_x, float focal_y,
	const float* colors,
	const float* transMats,
	const float4* normal_opacity,
	int* n_contrib,
	const float* bg_color,
	const float* mod_depth,
	float* out_color,
	float* out_others)
{
	renderCUDA<NUM_CHANNELS> << <grid, block >> > (
		ranges,
		point_list,
		W, H,
		focal_x, focal_y,
		colors,
		transMats,
		normal_opacity,
		n_contrib,
		bg_color,
		mod_depth,
		out_color,
		out_others);
}

void FORWARD::preprocess(int P, int D, int M,
	const float* means3D,
	const glm::vec2* scales,
	const float scale_modifier,
	const glm::vec4* rotations,
	const float* opacities,
	const float* shs,
	const float* transMat_precomp,
	const float* colors_precomp,
	const float* viewmatrix,
	const float* projmatrix,
	const glm::vec3* cam_pos,
	const int W, const int H,
	const float focal_x, const float focal_y,
	const float tan_fovx, const float tan_fovy,
	int* radii,
	float2* means2D,
	float* depths,
	float* transMats,
	float* rgb,
	float4* normal_opacity,
	const dim3 grid,
	uint32_t* tiles_touched,
	bool prefiltered)
{
	preprocessCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		scales,
		scale_modifier,
		rotations,
		opacities,
		shs,
		transMat_precomp,
		colors_precomp,
		viewmatrix, 
		projmatrix,
		cam_pos,
		W, H,
		tan_fovx, tan_fovy,
		focal_x, focal_y,
		radii,
		means2D,
		depths,
		transMats,
		rgb,
		normal_opacity,
		grid,
		tiles_touched,
		prefiltered
		);
}

void FORWARD::computeSH(int P, int D, int M,
	const float* means3D,
	const float* shs,
	const float* cam_pos,
	float* rgb)
{
	computeSHCUDA<NUM_CHANNELS> << <(P + 255) / 256, 256 >> > (
		P, D, M,
		means3D,
		shs,
		(const glm::vec3*)cam_pos,
		rgb
	);
}
